#include "hip/hip_runtime.h"
/* .cuda.cu - Copyright 2019/2020 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include ".cuda.h"

namespace lh2core
{

// path tracing buffers and global variables
__constant__ CoreInstanceDesc* instanceDescriptors;
__constant__ CUDAMaterial* materials;
__constant__ CoreLightTri* triLights;
__constant__ CorePointLight* pointLights;
__constant__ CoreSpotLight* spotLights;
__constant__ CoreDirectionalLight* directionalLights;
__constant__ int4 lightCounts;			// area, point, spot, directional
__constant__ uchar4* argb32;
__constant__ float4* argb128;
__constant__ uchar4* nrm32;
__constant__ float4* skyPixels;
__constant__ int skywidth;
__constant__ int skyheight;
__constant__ PathState* pathStates;
__constant__ float4* debugData;
__constant__ LightCluster* lightTree;

__constant__ mat4 worldToSky;

// path tracer settings
__constant__ __device__ float geometryEpsilon;
__constant__ __device__ float clampValue;

// staging: copies will be batched and carried out after rendering completes, 
// to allow the CPU to update the scene concurrently with GPU rendering.

enum { INSTS = 0, MATS, TLGHTS, PLGHTS, SLGHTS, DLGHTS, LCNTS, RGB32, RGBH, NRMLS, SKYPIX, SKYW, SKYH, SMAT, DBGDAT, GEPS, CLMPV, LTREE };

// device pointers are not real pointers for nvcc, so we need a bit of a hack.

struct StagedPtr { void* p; int id; };
struct StagedInt { int v; int id; };
struct StagedInt4 { int4 v; int id; };
struct StagedFloat3 { float3 v; int id; };
struct StagedMat { mat4 v; int id; };
struct StagedF32 { float v; int id; };
struct StagedCpy { void* d; void* s; int n; };
static std::vector<StagedPtr> stagedPtr;
static std::vector<StagedInt> stagedInt;
static std::vector<StagedInt4> stagedInt4;
static std::vector<StagedFloat3> stagedFloat3;
static std::vector<StagedMat> stagedMat;
static std::vector<StagedF32> stagedF32;
static std::vector<StagedCpy> stagedCpy;

__host__ static void pushPtrCpy( int id, void* p )
{
	if (id == INSTS) hipMemcpyToSymbol(HIP_SYMBOL( instanceDescriptors), &p, sizeof( void* ) );
	if (id == MATS) hipMemcpyToSymbol(HIP_SYMBOL( materials), &p, sizeof( void* ) );
	if (id == TLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( triLights), &p, sizeof( void* ) );
	if (id == PLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( pointLights), &p, sizeof( void* ) );
	if (id == SLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( spotLights), &p, sizeof( void* ) );
	if (id == DLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( directionalLights), &p, sizeof( void* ) );
	if (id == RGB32) hipMemcpyToSymbol(HIP_SYMBOL( argb32), &p, sizeof( void* ) );
	if (id == RGBH) hipMemcpyToSymbol(HIP_SYMBOL( argb128), &p, sizeof( void* ) );
	if (id == NRMLS) hipMemcpyToSymbol(HIP_SYMBOL( nrm32), &p, sizeof( void* ) );
	if (id == SKYPIX) hipMemcpyToSymbol(HIP_SYMBOL( skyPixels), &p, sizeof( void* ) );
	if (id == DBGDAT) hipMemcpyToSymbol(HIP_SYMBOL( debugData), &p, sizeof( void* ) );
	if (id == LTREE) hipMemcpyToSymbol(HIP_SYMBOL( lightTree), &p, sizeof( void* ) );
}
__host__ static void pushIntCpy( int id, const int v )
{
	if (id == SKYW) hipMemcpyToSymbol(HIP_SYMBOL( skywidth), &v, sizeof( int ) );
	if (id == SKYH) hipMemcpyToSymbol(HIP_SYMBOL( skyheight), &v, sizeof( int ) );
}
__host__ static void pushF32Cpy( int id, const float v )
{
	if (id == GEPS) hipMemcpyToSymbol(HIP_SYMBOL( geometryEpsilon), &v, sizeof( float ) );
	if (id == CLMPV) hipMemcpyToSymbol(HIP_SYMBOL( clampValue), &v, sizeof( int ) );
}
__host__ static void pushMatCpy( int id, const mat4& m )
{
	if (id == SMAT) hipMemcpyToSymbol(HIP_SYMBOL( worldToSky), &m, sizeof( mat4 ) );
}
__host__ static void pushInt4Cpy( int id, const int4& v )
{
	if (id == LCNTS) hipMemcpyToSymbol(HIP_SYMBOL( lightCounts), &v, sizeof( int4 ) );
}
__host__ static void pushFloat3Cpy( int id, const float3& v )
{
	// nothing here yet
}

#define MAXVARS	32
static void* prevPtr[MAXVARS] = {};
static int prevInt[MAXVARS] = {};
static float prevFloat[MAXVARS] = {};
static int4 prevInt4[MAXVARS] = {};
// static float3 prevFloat3[MAXVARS] = {};
static bool prevValSet[MAXVARS] = {};

__host__ static void stagePtrCpy( int id, void* p )
{
	if (prevPtr[id] == p) return; // not changed
	StagedPtr n = { p, id };
	stagedPtr.push_back( n );
	prevPtr[id] = p;
}
__host__ static void stageIntCpy( int id, const int v )
{
	if (prevValSet[id] == true && prevInt[id] == v) return;
	StagedInt n = { v, id };
	stagedInt.push_back( n );
	prevValSet[id] = true;
	prevInt[id] = v;
}
__host__ static void stageF32Cpy( int id, const float v )
{
	if (prevValSet[id] == true && prevFloat[id] == v) return;
	StagedF32 n = { v, id };
	stagedF32.push_back( n );
	prevValSet[id] = true;
	prevFloat[id] = v;
}
__host__ static void stageMatCpy( int id, const mat4& m ) { StagedMat n = { m, id }; stagedMat.push_back( n ); }
__host__ static void stageInt4Cpy( int id, const int4& v )
{
	if (prevValSet[id] == true && prevInt4[id].x == v.x && prevInt4[id].y == v.y && prevInt4[id].z == v.z && prevInt4[id].w == v.w) return;
	StagedInt4 n = { v, id };
	stagedInt4.push_back( n );
	prevValSet[id] = true;
	prevInt4[id] = v;
}
/* __host__ static void stageFloat3Cpy( int id, const float3& v )
{
	if (prevValSet[id] == true && prevFloat3[id].x == v.x && prevFloat3[id].y == v.y && prevFloat3[id].z == v.z) return;
	StagedFloat3 n = { v, id };
	stagedFloat3.push_back( n );
	prevValSet[id] = true;
	prevFloat3[id] = v;
} */

__host__ void stageMemcpy( void* d, void* s, int n ) { StagedCpy c = { d, s, n }; stagedCpy.push_back( c ); }

__host__ void stageInstanceDescriptors( CoreInstanceDesc* p ) { stagePtrCpy( INSTS /* instanceDescriptors */, p ); }
__host__ void stageMaterialList( CUDAMaterial* p ) { stagePtrCpy( MATS /* materials */, p ); }
__host__ void stageTriLights( CoreLightTri* p ) { stagePtrCpy( TLGHTS /* triLights */, p ); }
__host__ void stagePointLights( CorePointLight* p ) { stagePtrCpy( PLGHTS /* pointLights */, p ); }
__host__ void stageSpotLights( CoreSpotLight* p ) { stagePtrCpy( SLGHTS /* spotLights */, p ); }
__host__ void stageDirectionalLights( CoreDirectionalLight* p ) { stagePtrCpy( DLGHTS /* directionalLights */, p ); }
__host__ void stageARGB32Pixels( uint* p ) { stagePtrCpy( RGB32 /* argb32 */, p ); }
__host__ void stageARGB128Pixels( float4* p ) { stagePtrCpy( RGBH /* argb128 */, p ); }
__host__ void stageNRM32Pixels( uint* p ) { stagePtrCpy( NRMLS /* nrm32 */, p ); }
__host__ void stageSkyPixels( float4* p ) { stagePtrCpy( SKYPIX /* skyPixels */, p ); }
__host__ void stageSkySize( int w, int h ) { stageIntCpy( SKYW /* skywidth */, w ); stageIntCpy( SKYH /* skyheight */, h ); }
__host__ void stageWorldToSky( const mat4& worldToLight ) { stageMatCpy( SMAT /* worldToSky */, worldToLight ); }
__host__ void stageDebugData( float4* p ) { stagePtrCpy( DBGDAT /* debugData */, p ); }
__host__ void stageGeometryEpsilon( float e ) { stageF32Cpy( GEPS /* geometryEpsilon */, e ); }
__host__ void stageClampValue( float c ) { stageF32Cpy( CLMPV /* clampValue */, c ); }
__host__ void stageLightTree( LightCluster* t ) { stagePtrCpy( LTREE /* light tree */, t ); }
__host__ void stageLightCounts( int tri, int point, int spot, int directional )
{
	const int4 counts = make_int4( tri, point, spot, directional );
	stageInt4Cpy( LCNTS /* lightCounts */, counts );
}

__host__ void pushStagedCopies()
{
	for (auto c : stagedCpy) hipMemcpy( c.d, c.s, c.n, hipMemcpyHostToDevice ); stagedCpy.clear();
	for (auto n : stagedPtr) pushPtrCpy( n.id, n.p ); stagedPtr.clear();
	for (auto n : stagedInt) pushIntCpy( n.id, n.v ); stagedInt.clear();
	for (auto n : stagedInt4) pushInt4Cpy( n.id, n.v ); stagedInt4.clear();
	for (auto n : stagedFloat3) pushFloat3Cpy( n.id, n.v ); stagedFloat3.clear();
	for (auto n : stagedF32) pushF32Cpy( n.id, n.v ); stagedF32.clear();
	for (auto n : stagedMat) pushMatCpy( n.id, n.v ); stagedMat.clear();
}

// counters for persistent threads
static __device__ Counters* counters;
__global__ void InitCountersForExtend_Kernel( int pathCount )
{
	if (threadIdx.x != 0) return;
	counters->activePaths = pathCount;	// remaining active paths
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->generated = 0;			// persistent thread atomic for generate in .optix.cu
	counters->extensionRays = 0;		// compaction counter for extension rays
	counters->shadowRays = 0;			// compaction counter for connections
	counters->connected = 0;
	counters->totalExtensionRays = pathCount;
	counters->totalShadowRays = 0;
}
__host__ void InitCountersForExtend( int pathCount ) { InitCountersForExtend_Kernel << <1, 32 >> > (pathCount); }
__global__ void InitCountersSubsequent_Kernel()
{
	if (threadIdx.x != 0) return;
	counters->totalExtensionRays += counters->extensionRays;
	counters->activePaths = counters->extensionRays;	// remaining active paths
	counters->extended = 0;				// persistent thread atomic for genSecond in .optix.cu
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->extensionRays = 0;		// compaction counter for extension rays
}
__host__ void InitCountersSubsequent() { InitCountersSubsequent_Kernel << <1, 32 >> > (); }
__host__ void SetCounters( Counters* p ) { hipMemcpyToSymbol(HIP_SYMBOL( counters), &p, sizeof( void* ) ); }

// functional blocks
#include "tools_shared.h"
#include "sampling_shared.h"
#include "material_shared.h"
#include "lights_shared.h"
#include "bsdf.h"
#include "pathtracer.h"
#include "finalize_shared.h"

} // namespace lh2core

// EOF